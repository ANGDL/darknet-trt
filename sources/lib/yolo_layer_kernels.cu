#include "hip/hip_runtime.h"
﻿#include <assert.h>
#include "yolo_layer_kernels.cuh"


__device__
float d_sigmod(float x) {
	return 1.0f / (1.0 + __expf(x));
}

__global__
void kernel_yolo_layer(
	const float* input,
	float* output,
	unsigned int grid_size,
	unsigned int num_classes,
	unsigned int num_boxes,
	unsigned int output_size
) {
	unsigned int id_x = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int id_y = threadIdx.y + blockDim.y * blockIdx.y;
	unsigned int id_z = threadIdx.z + blockDim.z * blockIdx.z;

	if (id_x >= grid_size || id_y >= grid_size || id_z >= num_boxes) {
		return;
	}

	unsigned int num_grids = grid_size * grid_size;
	unsigned int grid_idx = id_x + grid_size * id_y;

	unsigned int loc_idx = grid_idx + num_grids * (id_z * (num_classes + 5));
	output[loc_idx + 0] = d_sigmod(input[loc_idx + 0]);  // sigmod(tx)
	output[loc_idx + 1] = d_sigmod(input[loc_idx + 1]);  // sigmod(ty)
	output[loc_idx + 2] = __expf(input[loc_idx + 2]);  // exp(tw)
	output[loc_idx + 3] = __expf(input[loc_idx + 3]);  // exp(th)
	output[loc_idx + 4] = d_sigmod(input[loc_idx + 4]);  // sigmod(to)

	for (cuuint32_t i = 0; i < num_boxes; ++i) {
		output[loc_idx + 5 + i] = d_sigmod(input[loc_idx + 5 + i]);  // confidence score
	}
}


hipError_t cuda_yolo_layer(
	const void* input,
	void* output,
	int batch_size,
	unsigned int grid_size,
	unsigned int num_classes,
	unsigned int num_boxes,
	unsigned int output_size,
	hipStream_t stream
) {
	assert(num_boxes == 3);
	dim3 threads_per_blocks(16, 16, 4);
	dim3 num_blocks(grid_size / threads_per_blocks.x, grid_size / threads_per_blocks.y, 1);

	const float* input_f = reinterpret_cast<const float*>(input);
	float* output_f = reinterpret_cast<float*>(output);

	for (int i = 0; i < batch_size; ++i) {
		kernel_yolo_layer << <num_blocks, threads_per_blocks >> > (
			input_f, output_f, grid_size, num_classes, num_boxes, output_size);
	}

	return hipGetLastError();
}


// kernel_upsample
// reference: https://github.com/pjreddie/darknet/blob/master/src/blas_kernels.cu
__global__ void kernel_upsample(size_t N, float* x, int w, int h, int c, int batch, int stride, float scale, float* out)
{
	size_t i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= N) return;
	int out_index = i;
	int out_w = i % (w * stride);
	i = i / (w * stride);
	int out_h = i % (h * stride);
	i = i / (h * stride);
	int out_c = i % c;
	i = i / c;
	int b = i % batch;

	int in_w = out_w / stride;
	int in_h = out_h / stride;
	int in_c = out_c;

	int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

	out[out_index] += scale * x[in_index];
}


hipError_t cuda_upsample_layer(const void* input, void* output, int batch_size, float stride,
	int c, int h, int w, hipStream_t stream)
{
	unsigned int size = w * h * c * batch_size * stride * stride;
	kernel_upsample << <cuda_gridsize(size), KERNEL_BLOCK >> > (size, (float*)input, w, h, c, batch_size, stride, 1.0, (float*)output);
	return hipGetLastError();
}
