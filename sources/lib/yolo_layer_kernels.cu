#include "hip/hip_runtime.h"
﻿#include <assert.h>
#include "yolo_layer_kernels.cuh"


__device__
float d_sigmod(float x) {
	return 1.0f / (1.0 + __expf(x));
}

__global__
void kernel_yolo_layer(
	const float* input,
	float* output,
	unsigned int grid_size,
	unsigned int num_classes,
	unsigned int num_boxes,
	unsigned int output_size
) {
	unsigned int id_x = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int id_y = threadIdx.y + blockDim.y * blockIdx.y;
	unsigned int id_z = threadIdx.z + blockDim.z * blockIdx.z;

	if (id_x >= grid_size || id_y >= grid_size || id_z >= num_boxes) {
		return;
	}

	unsigned int num_grids = grid_size * grid_size;
	unsigned int grid_idx = id_x + grid_size * id_y;

	unsigned int loc_idx = grid_idx + num_grids * (id_z * (num_classes + 5));
	output[loc_idx + 0] = d_sigmod(input[loc_idx + 0]);  // sigmod(tx)
	output[loc_idx + 1] = d_sigmod(input[loc_idx + 1]);  // sigmod(ty)
	output[loc_idx + 2] = __expf(input[loc_idx + 2]);  // exp(tw)
	output[loc_idx + 3] = __expf(input[loc_idx + 3]);  // exp(th)
	output[loc_idx + 4] = d_sigmod(input[loc_idx + 4]);  // sigmod(to)

	for (cuuint32_t i = 0; i < num_boxes; ++i) {
		output[loc_idx + 5 + i] = d_sigmod(input[loc_idx + 5 + i]);  // confidence score
	}
}


hipError_t cuda_yolo_layer(
	const void* input,
	void* output,
	int batch_size,
	unsigned int grid_size,
	unsigned int num_classes,
	unsigned int num_boxes,
	unsigned int output_size,
	hipStream_t stream
) {
	assert(num_boxes == 3);
	dim3 threads_per_blocks(16, 16, num_boxes);
	dim3 num_blocks(grid_size / threads_per_blocks.x, grid_size / threads_per_blocks.y, 1);

	const float* input_f = reinterpret_cast<const float*>(input);
	float* output_f = reinterpret_cast<float*>(output);

	for (int i = 0; i < batch_size; ++i) {
		kernel_yolo_layer << <num_blocks, threads_per_blocks >> > (
			input_f, output_f, grid_size, num_classes, num_boxes, output_size);
	}

	return hipGetLastError();
}
